#include <iostream>
#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <stdio.h>

const double PI = 3.141592653589793238460;

int main() {
	
	int N = 10;
	size_t memorySize = N * sizeof(hipfftComplex);

	hipfftComplex* A = (hipfftComplex*) malloc(memorySize);
	hipfftComplex* B = (hipfftComplex*)malloc(memorySize);

	for (int i = 0; i < N; i++) {
		A[i].x = (float) sin(2 * PI * i / N);
		A[i].y = 0.0;
	}

	hipfftComplex* d_A;
	hipfftComplex* d_B;

	hipMalloc(&d_A, memorySize);
	hipMalloc(&d_B, memorySize);

	hipMemcpy(d_A, A, memorySize, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, N, HIPFFT_C2C, 1);
	hipfftExecC2C(plan, d_A, d_B, HIPFFT_FORWARD);

	hipMemcpy(B, d_B, memorySize, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		printf("%f %f %f %f\n", A[i].x, A[i].y, B[i].x, B[i].y);
	}

	return 0;
}